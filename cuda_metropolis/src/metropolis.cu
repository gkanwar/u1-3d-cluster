#include "hip/hip_runtime.h"
#ifndef METROPOLIS_KERNEL_CU
#define METROPOLIS_KERNEL_CU

#include <cassert>
#include <cstdio>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "config.h"
#include "util.h"

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n", file, line,
            (int)err, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

__device__ double sq(double x) {
  return x*x;
}

__device__ double get_rand_float(hiprandState* thread_state) {
  return hiprand_uniform(thread_state);
}
__device__ int get_rand_int(hiprandState* thread_state, int max) {
  return (int) max * (1.0 - hiprand_uniform(thread_state));
}


__device__ int get_thread_id() {
  int blockId = blockIdx.x + blockIdx.y * gridDim.x
      + blockIdx.z * gridDim.x * gridDim.y;
  int threadId = blockId * blockDim.x * blockDim.y * blockDim.z
      + threadIdx.z * blockDim.x * blockDim.y
      + threadIdx.y * blockDim.x + threadIdx.x;
  return threadId;
}

__global__ void init_curand(hiprandState* state, unsigned long seed) {
  int idx = get_thread_id();
  hiprand_init(seed, idx, 0, &state[idx]);
}

__device__ inline unsigned get_idx(unsigned x, unsigned y, unsigned z, unsigned L) {
  return x*L*L + y*L + z;
}

__device__ double get_local_action(
    double h, double h_xp, double h_xm, double h_yp, double h_ym, double h_zp,
    double h_zm, double e2) {
  return (e2/2.0) * (
      sq(h - h_xp) +
      sq(h - h_xm) +
      sq(h - h_yp) +
      sq(h - h_ym) +
      sq(h - h_zp) +
      sq(h - h_zm) );
}

__global__ void metropolis_kernel(int* cfg, hiprandState* rng_state, double e2, int parity, unsigned L) {
  // thread index has x running fastest, z slowest, whereas our coordinates have
  // x running slowest, z fastest... so we swap labels at this point to match
  const unsigned int z0 = (blockIdx.x * blockDim.x + threadIdx.x) * THREAD_L;
  const unsigned int y0 = (blockIdx.y * blockDim.y + threadIdx.y) * THREAD_L;
  const unsigned int x0 = (blockIdx.z * blockDim.z + threadIdx.z) * THREAD_L;
  const int threadId = get_thread_id();
  hiprandState* thread_rng_state = &rng_state[threadId];
  for (unsigned int x = x0; x < x0+THREAD_L; ++x) {
    for (unsigned int y = y0; y < y0+THREAD_L; ++y) {
      // NOTE: we assume THREAD_L is even, so (x0,y0,z0) is an EVEN
      // site, meaning we can start at (x0,y0,z0+parity) and advance by 2.
      for (unsigned int z = z0+parity; z < z0+THREAD_L; z += 2) {
        int cfg_site = cfg[get_idx(x, y, z, L)];
        const double h_xp = cfg[get_idx((x+1)%L, y, z, L)] / 2.0;
        const double h_xm = cfg[get_idx((x+L-1)%L, y, z, L)] / 2.0;
        const double h_yp = cfg[get_idx(x, (y+1)%L, z, L)] / 2.0;
        const double h_ym = cfg[get_idx(x, (y+L-1)%L, z, L)] / 2.0;
        const double h_zp = cfg[get_idx(x, y, (z+1)%L, L)] / 2.0;
        const double h_zm = cfg[get_idx(x, y, (z+L-1)%L, L)] / 2.0;
        double S = get_local_action(cfg_site/2.0, h_xp, h_xm, h_yp, h_ym, h_zp, h_zm, e2);
        for (unsigned i = 0; i < N_METROPOLIS_HITS; ++i) {
          const int dcfg = 4*get_rand_int(thread_rng_state, 2) - 2;
          const double new_S = get_local_action(
              (cfg_site+dcfg)/2.0, h_xp, h_xm, h_yp, h_ym, h_zp, h_zm, e2);
          if (get_rand_float(thread_rng_state) < exp(-new_S + S)) {
            cfg_site += dcfg;
            S = new_S;
          }
        }
        cfg[get_idx(x, y, z, L)] = cfg_site;
      }
    }
  }
}

__global__ void measure_E(const int* cfg, double* E, unsigned L) {
  const unsigned int z0 = (blockIdx.x * blockDim.x + threadIdx.x) * THREAD_L;
  const unsigned int y0 = (blockIdx.y * blockDim.y + threadIdx.y) * THREAD_L;
  const unsigned int x0 = (blockIdx.z * blockDim.z + threadIdx.z) * THREAD_L;
  for (unsigned int x = x0; x < x0+THREAD_L; ++x) {
    for (unsigned int y = y0; y < y0+THREAD_L; ++y) {
      for (unsigned int z = z0; z < z0+THREAD_L; ++z) {
        const double h = cfg[get_idx(x, y, z, L)] / 2.0;
        const double h_xp = cfg[get_idx((x+1)%L, y, z, L)] / 2.0;
        const double h_yp = cfg[get_idx(x, (y+1)%L, z, L)] / 2.0;
        const double h_zp = cfg[get_idx(x, y, (z+1)%L, L)] / 2.0;
        E[get_idx(x, y, z, L)] = (
            sq(h - h_xp) +
            sq(h - h_yp) +
            sq(h - h_zp) );
      }
    }
  }
}

__global__ void measure_OC(const int* cfg, double* OC, unsigned L) {
  const unsigned int z0 = (blockIdx.x * blockDim.x + threadIdx.x) * THREAD_L;
  const unsigned int y0 = (blockIdx.y * blockDim.y + threadIdx.y) * THREAD_L;
  const unsigned int x0 = (blockIdx.z * blockDim.z + threadIdx.z) * THREAD_L;
  for (unsigned int x = x0; x < x0+THREAD_L; ++x) {
    for (unsigned int y = y0; y < y0+THREAD_L; ++y) {
      for (unsigned int z = z0; z < z0+THREAD_L; ++z) {
        const double h = cfg[get_idx(x, y, z, L)] / 2.0;
        const double h1 = cfg[get_idx((x+1)%L, y, z, L)] / 2.0;
        const double h2 = cfg[get_idx(x, (y+1)%L, z, L)] / 2.0;
        const double h3 = cfg[get_idx(x, y, (z+1)%L, L)] / 2.0;
        const double h12 = cfg[get_idx((x+1)%L, (y+1)%L, z, L)] / 2.0;
        const double h23 = cfg[get_idx(x, (y+1)%L, (z+1)%L, L)] / 2.0;
        const double h13 = cfg[get_idx((x+1)%L, y, (z+1)%L, L)] / 2.0;
        const double h123 = cfg[get_idx((x+1)%L, (y+1)%L, (z+1)%L, L)] / 2.0;
        const double h_bar = (h + h1 + h2 + h3 + h12 + h23 + h13 + h123) / 8.0;
        OC[get_idx(x, y, z, L)] = (
            sq(h - h_bar) + sq(h12 - h_bar) +
            sq(h23 - h_bar) + sq(h13 - h_bar)
            - sq(h1 - h_bar) - sq(h2 - h_bar)
            - sq(h3 - h_bar) - sq(h123 - h_bar) );
      }
    }
  }
}

__global__ void init_cfg(int* cfg, unsigned L) {
  const unsigned int z0 = (blockIdx.x * blockDim.x + threadIdx.x) * THREAD_L;
  const unsigned int y0 = (blockIdx.y * blockDim.y + threadIdx.y) * THREAD_L;
  const unsigned int x0 = (blockIdx.z * blockDim.z + threadIdx.z) * THREAD_L;
  for (unsigned int x = x0; x < x0+THREAD_L; ++x) {
    for (unsigned int y = y0; y < y0+THREAD_L; ++y) {
      for (unsigned int z = z0; z < z0+THREAD_L; ++z) {
        cfg[get_idx(x, y, z, L)] = (x+y+z) % 2;
      }
    }
  }
}

extern "C" int* alloc_and_init_cfg(int L, dim3 grid_shape, dim3 block_shape) {
  int* d_cfg = NULL;
  checkCudaErrors(hipMalloc((void**)&d_cfg, L*L*L*sizeof(int)));
  init_cfg<<<grid_shape, block_shape>>>(d_cfg, L);
  return d_cfg;
}

extern "C" hiprandState* init_rng(unsigned long seed, dim3 grid_shape, dim3 block_shape) {
  hiprandState* rng_state = NULL;
  int block_size = block_shape.x * block_shape.y * block_shape.z;
  int grid_size = grid_shape.x * grid_shape.y * grid_shape.z;
  checkCudaErrors(hipMalloc(
      (void**)&rng_state,
      block_size * grid_size * sizeof(hiprandState)));
  init_curand<<<grid_shape, block_shape>>>(rng_state, seed);
  return rng_state;
}

extern "C" void free_cfg(int* d_cfg) {
  if (d_cfg) {
    checkCudaErrors(hipFree(d_cfg));
  }
}

extern "C" void free_rng(hiprandState* rng_state) {
  if (rng_state) {
    checkCudaErrors(hipFree(rng_state));
  }
}

void copy_dev_to_host(double* dest, const double* d_src, unsigned n) {
  checkCudaErrors(hipMemcpy(
      (void*)dest, (void*)d_src, n*sizeof(double),
      hipMemcpyDeviceToHost));
}

extern "C" void run_metropolis(
    int* d_cfg, double e2, int L, int n_iter, int n_therm, int n_skip_meas,
    dim3 grid_shape, dim3 block_shape, hiprandState* rng_state,
    double* E_hist, double* MC_hist) {

  assert(L == grid_shape.x * block_shape.x * THREAD_L);
  assert(L == grid_shape.y * block_shape.y * THREAD_L);
  assert(L == grid_shape.z * block_shape.z * THREAD_L);
  double* d_tmp_E = NULL;
  double* d_tmp_OC = NULL;
  checkCudaErrors(hipMalloc((void**)&d_tmp_E, L*L*L*sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&d_tmp_OC, L*L*L*sizeof(double)));
  double* tmp_E =  (double*) malloc(L*L*L*sizeof(double));
  double* tmp_OC =  (double*) malloc(L*L*L*sizeof(double));

  for (int i = -n_therm; i < n_iter; ++i) {

    if ((i+1) % 1000 == 0) {
      printf("%d / %d\n", i+1, n_iter);
    }

    metropolis_kernel<<<grid_shape, block_shape>>>(d_cfg, rng_state, e2, 0, L);
    metropolis_kernel<<<grid_shape, block_shape>>>(d_cfg, rng_state, e2, 1, L);

    if (i >= 0 && (i+1) % n_skip_meas == 0) {
      // measure all arrays on device
      measure_E<<<grid_shape, block_shape>>>(d_cfg, d_tmp_E, L);
      measure_OC<<<grid_shape, block_shape>>>(d_cfg, d_tmp_OC, L);
      checkCudaErrors(hipDeviceSynchronize());

      // copy to host and reduce
      copy_dev_to_host(tmp_E, d_tmp_E, L*L*L);
      copy_dev_to_host(tmp_OC, d_tmp_OC, L*L*L);
      double E = sum_field(tmp_E, L);
      double MC = sum_field_staggered(tmp_OC, L);

      // log history
      int meas_ind = ((i+1) / n_skip_meas) - 1;
      E_hist[meas_ind] = E;
      MC_hist[meas_ind] = MC;
    }
  }

  free(tmp_E);
  free(tmp_OC);
  checkCudaErrors(hipFree(d_tmp_E));
  checkCudaErrors(hipFree(d_tmp_OC));
}


#endif
